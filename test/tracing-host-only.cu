#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include "../lib/Common.h"

extern "C" {
void __trace_touch(hipStream_t stream);
void __trace_start(hipStream_t stream, const char *kernel_name);
void __trace_fill_info(const void *info, hipStream_t stream);
void __trace_copy_to_symbol(hipStream_t stream, const char* symbol, const void *info);
void __trace_stop(hipStream_t stream);
}

#define cudaChecked(code) do {\
  hipError_t err = code;\
  if (err != hipSuccess) {\
    printf("CUDA error at %s:%d: %s\n", __FILE__, __LINE__,\
        hipGetErrorString(err));\
    abort();\
  }\
} while(0)\

void add_trace(uint32_t *fronts, uint32_t *backs, uint64_t *traces,
    int slot, uint64_t desc, uint64_t addr, uint64_t size) {

  while (fronts[slot] >= SLOTS_SIZE) {}

  size_t offset = slot * SLOTS_SIZE + fronts[slot]*3;
  fronts[slot]++;
  traces[offset + 0] = desc;
  traces[offset + 1] = addr;
  traces[offset + 2] = size;
  backs[slot]++;
}

int main(int argc, char** argv) {
  // check if unified addressing is used, so that cudaHostGetDevicePtr is the
  // identity function.
  hipDeviceProp_t prop;
  cudaChecked(hipGetDeviceProperties(&prop, 0));

  if (!prop.unifiedAddressing) {
    printf("unified addressing not supported, unable to test device access from host\n");
    exit(0);
  }

  setenv("MEMTRACE_PATTERN", "./test-trace-host", 1);
  __trace_touch(NULL);
  printf("starting trace\n");
  __trace_start(NULL, "test");
  traceinfo_t info;
  __trace_fill_info(&info, NULL);

  uint32_t *fronts = info.front;
  uint32_t *backs = info.back;
  uint64_t *traces = info.slot;

  for (int i = 0; i < 256; ++i) {
    add_trace(fronts, backs, traces, 0, 3*i, 3*i + 1, 3*i + 2);
  }

  printf("stopping trace\n");
  __trace_stop(NULL);
  cudaChecked(hipStreamSynchronize(NULL));

  return 0;
}
