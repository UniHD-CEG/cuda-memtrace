#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "../lib/Common.h"

#define cudaChecked(code) do {\
  hipError_t err = code;\
  if (err != hipSuccess) {\
    printf("CUDA error at %s:%d: %s\n", __FILE__, __LINE__,\
        hipGetErrorString(err));\
    abort();\
  }\
} while(0)\

extern "C" {
void __trace_touch(hipStream_t stream);
void __trace_start(hipStream_t stream, const char *kernel_name);
void __trace_fill_info(const void *info, hipStream_t stream);
void __trace_copy_to_symbol(hipStream_t stream, const char* symbol, const void *info);
void __trace_stop(hipStream_t stream);
}

extern "C"
__device__ void __mem_trace (uint8_t* records, uint8_t* allocs, uint8_t* commits,
        uint64_t desc, uint64_t addr, uint32_t slot) {
    uint64_t cta = blockIdx.x;
    cta <<= 16;
    cta |= blockIdx.y;
    cta <<= 16;
    cta |= blockIdx.z;

    uint32_t lane_id;
    asm volatile ("mov.u32 %0, %%laneid;" : "=r"(lane_id));

    uint32_t active   = __ballot(1); // get number of active threads 
    uint32_t rlane_id = __popc(active << (32 - lane_id));
    uint32_t n_active = __popc(active);
    uint32_t lowest   = __ffs(active)-1;

    uint32_t *alloc = (uint32_t*)(&allocs[slot * CACHELINE]);
    uint32_t *commit = (uint32_t*)(&commits[slot * CACHELINE]);

    volatile uint32_t *valloc = alloc;
    volatile uint32_t *vcommit = commit;
    unsigned int id = 0;

    if (lane_id == lowest) {
      while(*valloc > (SLOTS_SIZE - 32) || (id = atomicAdd(alloc, n_active)) > (SLOTS_SIZE - 32)) {
        (void)0;
      }
    }

    uint32_t slot_offset = slot * SLOTS_SIZE;
    uint32_t record_offset = __shfl(id, lowest) + rlane_id;
    record_t *record = (record_t*) &(records[(slot_offset + record_offset) * RECORD_SIZE]);
    record->desc = desc;
    record->addr = addr;
    record->cta  = cta;
    __threadfence_system(); 

    if (lane_id == lowest ) atomicAdd(commit, n_active);
}

__global__ void test_kernel(uint8_t* records, uint8_t* allocs, uint8_t* commits, int n) {
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid % 2 == 0)
    return;
  for (int i = 0; i < n; ++i) {
    __mem_trace(records, allocs, commits, gid, gid/32, gid/32 % SLOTS_NUM);
  }
}

int main(int argc, char** argv) {
  // check if unified addressing is used, so that cudaHostGetDevicePtr is the
  // identity function.
  hipDeviceProp_t prop;
  cudaChecked(hipGetDeviceProperties(&prop, 0));

  const char* rounds_str = getenv("ROUNDS");
  int32_t rounds = rounds_str ? strtol(rounds_str, NULL, 10) : 1;

  const char* threads_str = getenv("THREADS");
  int32_t threads = threads_str ? strtol(threads_str, NULL, 10) : 32;

  if (!prop.unifiedAddressing) {
    printf("unified addressing not supported, unable to test device access from host\n");
    exit(0);
  }

  printf("trace for threads with gid.x %% 2 != 0\n");
  printf("%d threads, %d rounds, expect %d records\n", threads, rounds, threads * rounds / 2);

  setenv("MEMTRACE_PATTERN", "./test-trace-device", 1);
  __trace_touch(NULL);
  printf("starting trace\n");
  __trace_start(NULL, "test");

  traceinfo_t info;
  __trace_fill_info(&info, NULL);

  uint8_t *allocs = info.allocs;
  uint8_t *commits = info.commits;
  uint8_t *records = info.records;
  test_kernel<<<1, threads>>>(records, allocs, commits, rounds);
  cudaChecked(hipDeviceSynchronize());

  printf("stopping trace\n");
  __trace_stop(NULL);
  cudaChecked(hipStreamSynchronize(NULL));
  cudaChecked(hipDeviceSynchronize());

  return 0;
}
